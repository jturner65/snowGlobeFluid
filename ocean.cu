///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}

// complex math functions
__device__
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__
float2 complex_add(float2 a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__
float2 interp2F2(float2 a, float2 b, float d)
{
	return make_float2(a.x + d*(b.x-a.x), a.y + d*(b.y-a.y)); 
}
__device__
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

//convert passed list of frequencies to appropriate array of float2
extern "C"
__global__ void buildFrequencyDataKernel(float2* freq_out,
										float* freq_rList,						//single dimension array of 1024 elements
										float* freq_cList,
                                       	unsigned int in_width,
                                       	unsigned int out_width,
                                       	unsigned int out_height,
										unsigned int is_NoteFreqs, 
										float thresh,
										float t)				//1 if notes, 0 if audio
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    
    unsigned int in_index = y*in_width+x;
    unsigned int out_index = y*out_width+x;
    
    unsigned int inx = (x % (in_width-1))+1;
	unsigned int iny = (y % (in_width-1))+1;
//  unsigned int inx = in_width- (x % (in_width-1));
//	unsigned int iny = in_width- (y % (in_width-1));

    float u = x / (float) out_width;
    float v = y / (float) out_height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;
    
	float scFct = .1f;
	t = t+scFct;
//	unsigned int totalOut = out_width * out_height;
//	unsigned int colOff = out_width/2;
//	unsigned int rowOff = (out_width * colOff);
//	unsigned int newIdx = (rowOff + (out_width*(out_index+colOff)/out_width) + 			
//				((colOff + (out_index%out_width)) % out_height))%totalOut;
	//if note frequencies, get complex version of note data, otherwise use freq_rList and freq_cList
	//e^j2pifot = cos(2pifot)<---freq_rList from audio + j(sin(2pifot) <---freq_cList from audio)    
	
//	if(is_NoteFreqs == 0){
		if ((x < out_width) && (y < out_height)) { 	//in_width == out_width
//			float freqR = logf(1 +(freq_rList[inx] < thresh ? thresh : freq_rList[inx]))-1;
//			float freqC = logf(1 +(freq_cList[iny] < thresh ? thresh : freq_cList[iny]))-1;
			float freqR = (freq_rList[inx] < thresh ? thresh : freq_rList[inx]);
			float freqC = (freq_cList[iny] < thresh ? thresh : freq_cList[iny]);
			
			freqR = freqR / powf(2,llrintf(log2f(freqR+1))-1);
			freqC = freqC / powf(2,llrintf(log2f(freqC+1))-1);
			
//			freq_out[out_index] = make_float2(sinf(u*freq + t) * cosf(v*freq + t) * scFct, sinf(v*freq + t) * cosf(u*freq + t) * scFct);
			freq_out[out_index] = make_float2(sinf(u*freqR + t) * cosf(v*freqR + t) * scFct, sinf(v*freqC + t) * cosf(u*freqC + t) * scFct);
	    	//freq_out[out_index] = make_float2(freqR *scFct, freqC *scFct);
	    	//freq_out[newIdx] = make_float2(freqR * scFct, freqC * scFct);
	    	//freq_out[newIdx] = make_float2(sinf(u*freqR + t) * cosf(v*freqC + t) * scFct, sinf(v*freqR + t) * cosf(u*freqC + t) * scFct);
		}
	
//	} else {
//		if ((x < out_width) && (y < out_height)) { 	//need to send in FFT!
//			float freqR = (freq_rList[inx] < thresh ? thresh : freq_rList[inx]);
//			float freqC = (freq_cList[iny] < thresh ? thresh : freq_cList[iny]);
//			freqR = freqR / powf(2,llrintf(log2f(freqR+1))-1);
//			freqC = freqC / powf(2,llrintf(log2f(freqC+1))-1);
//			freq_out[out_index] = make_float2(sinf(u*freqR + t) * cosf(v*freqR + t) * scFct, sinf(v*freqC + t) * cosf(u*freqC + t) * scFct);
//		}
//	}
//	//freq_out[out_index] 

}
// generate wave heightfield at time t based on initial heightfield and dispersion relationship
extern "C"
__global__ void generateSpectrumKernel(float2* h0, float2* ht,float2* freq, unsigned int in_width, unsigned int out_width, unsigned int out_height,
                                       float t,float mix,float patchSize)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;
    
    // calculate wave vector
    float2 k;
    float twoPiInvPtch = (2.0f * HIP_PI_F / patchSize);
    k.x = (-(int)out_width / 2.0f + x) * twoPiInvPtch;
    k.y = (-(int)out_height / 2.0f + y) * twoPiInvPtch;

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);

	if ((x < out_width) && (y < out_height)) {
		float2 h0_k = h0[in_index];
		float2 h0_mk = h0[in_mindex];
		float2 tmpRes1 = complex_add( complex_mult(h0_k, complex_exp(w * t)), complex_mult(conjugate(h0_mk), complex_exp(-w * t)) );
		//float2 tmpRes2 = make_float2 (freq[out_index].x + tmpRes1.x,freq[out_index].y + tmpRes1.y);
		float2 tmpRes2 = freq[out_index];
		
        // output frequency-space complex values
		//ht[out_index] = complex_add( complex_mult(h0_k, complex_exp(w * t)), complex_mult(conjugate(h0_mk), complex_exp(-w * t)) );
		ht[out_index] = interp2F2(tmpRes1,tmpRes2,mix);
	}
}

// update height map values based on output of FFT
extern "C"
__global__ void updateHeightmapKernel(float*  heightMap, float2* ht, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;
    
    float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;
	heightMap[i] = ht[i].x * sign_correction;
}

// generate slope by partial differences in spatial domain
extern "C"
__global__ void calculateSlopeKernel(float* h, float2 *slopeOut, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float2 slope = make_float2(0.0f, 0.0f);
    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1)) {
        slope.x = h[i+1] - h[i-1];
        slope.y = h[i+width] - h[i-width];
    }
    slopeOut[i] = slope;
}
